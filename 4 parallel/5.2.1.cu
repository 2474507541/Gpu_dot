#include "hip/hip_runtime.h"
#include <iostream>
#include "hip/hip_runtime.h";
#include ""

#define N (66 * 1024)

__global__ void add(int* a, int* b, int* c)
{
	//ת����������
	int tid = threadIdx.x + blockIdx.x * blockDim.x;

	while (tid < N)
	{
		c[tid] = a[tid] + b[tid];
		tid += blockDim.x * gridDim.x;
	}
}

int main()
{
	int a[N], b[N], c[N];
	int* dev_a, * dev_b, * dev_c;

	hipMalloc((void**)&dev_a, N * sizeof(int));
	hipMalloc((void**)&dev_b, N * sizeof(int));
	hipMalloc((void**)&dev_c, N * sizeof(int));

	for (int i = 0; i < N; i++)
	{
		a[i] = -i;
		b[i] = i * i;
	}

	hipMemcpy(dev_a, a, N * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_b, b, N * sizeof(int), hipMemcpyHostToDevice);

	add << <128, 128 >> > (dev_a, dev_b, dev_c);

	hipMemcpy(c, dev_c, N * sizeof(int), hipMemcpyDeviceToHost);

	bool success = true;
	for (int i = 0; i < N; i++)
	{
		if (a[i] + b[i] != c[i])
		{
			printf("%d + %d != %d\n", a[i], b[i], c[i]);
			success = false;
		}
	}

	if (success) printf("Did it.");

	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);

	return 0;
}